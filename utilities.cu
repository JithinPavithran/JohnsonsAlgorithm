#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "graph.h"
#include "utilities.h"
#include <stdio.h>
#include <stdlib.h>

__host__ __device__ int BLOCKS(int n){
 	return (int)ceil(((double)(n))/1024);
}

__host__ __device__ int THREADS(int n){
	return n > 1024 ? 1024 : n;
}

__host__ void checkForErr(char* str){
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if(err!=0){
		printf(RED "ERROR: %s\n" RESET, str);
		printf(RED "ERROR=%d, %s, %s\n" RESET, err, hipGetErrorName(err), 
			hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	else{
		printf(GRN "SUCCESS: %s\n" RESET, str);
	}
}

void copyToDevice(Node **nodesD, Edge **edgesD,
				 Node **nodesH, Edge **edgesH,
				 int nodes_n, int edges_n){
	hipMalloc(nodesD, sizeof(Node)*nodes_n);
	hipMalloc(edgesD, sizeof(Edge)*edges_n);
	hipMemcpy(*nodesD, *nodesH, sizeof(Node)*nodes_n,
				hipMemcpyHostToDevice);
	hipMemcpy(*edgesD, *edgesH, sizeof(Edge)*edges_n,
				hipMemcpyHostToDevice);
}

void copyNodesToHost(Node **nodesD, Node **nodesH, int nodes_n){
	hipMemcpy(*nodesH, *nodesD, sizeof(Node)*(nodes_n),
				hipMemcpyDeviceToHost);
}

void copyEdgesToDevice(Edge **edgesD, Edge **edgesH, int edges_n){
	hipMemcpy(*edgesD, *edgesH, sizeof(Edge)*(edges_n),
				hipMemcpyHostToDevice);
}

void copyToHost(Node **nodesD, Edge **edgesD,
				 Node **nodesH, Edge **edgesH,
				 int nodes_n, int edges_n){
	hipMemcpy(*nodesH, *nodesD, sizeof(Node)*nodes_n,
				hipMemcpyDeviceToHost);
	hipMemcpy(*edgesH, *edgesD, sizeof(Edge)*edges_n,
				hipMemcpyDeviceToHost);
}
